#include "hip/hip_runtime.h"
#include "volume_renderer.hpp"
#include "optics_data.hpp"
#include "config.hpp"
#include "utils.hpp"


/**
 * Kernels for ray casting and image composition
 */

__device__ static void compositeFrontToBack(
    Eigen::Vector3f& color_dst, Eigen::Vector3f& alpha_dst,
    Eigen::Vector3f color_src, Eigen::Vector3f alpha_src)
{
    color_dst = color_dst + (Eigen::Vector3f::Ones() - alpha_dst).cwiseProduct(color_src);
    alpha_dst = (alpha_dst + (Eigen::Vector3f::Ones() - alpha_dst).cwiseProduct(alpha_src)).cwiseMin(1.0).cwiseMax(0.0);
}

__global__ static void rayIntegral(
    Eigen::Vector3f* pixel_array, ImplicitGeometry** geom, Camera** cam,
    Light** lis, int lis_num, Classifier** cls, float ambient, float shininess, float dt)
{
    int max_x = (*cam)->getFilm().resolution.x();
    int max_y = (*cam)->getFilm().resolution.y();

    int dx = blockIdx.x * blockDim.x + threadIdx.x;
    int dy = blockIdx.y * blockDim.y + threadIdx.y;

    if((dx >= max_x) || (dy >= max_y))
    {
        return;
    }

    Eigen::Vector3f color(0, 0, 0);
    Eigen::Vector3f alpha(0, 0, 0);
    Ray ray = (*cam)->generateRay((float)dx, (float)dy);
    float t_start = 0, t_end = 0;

    /* Integration calculation */
    if ((*geom)->bboxRayIntersection(ray, t_start, t_end))
    {
        float t = t_start;
        while (t <= t_end)
        {
            /* Get position of the sampled point */
            Eigen::Vector3f p = ray.getPoint(t);
            /* Sample the point on the geometry */
            VolumeSampleData pt_data = (*geom)->sample(p);

            /* Get optical data by transfer function */
            OpticsData opt_d = (*cls)->transfer(pt_data, *cam, lis, lis_num, ambient, shininess, dt);

            /* Front-to-back composition */
            compositeFrontToBack(color, alpha, opt_d.getColor(), opt_d.getOpacity());

            /* Early ray termination */
            if (alpha.x() >= 1.0 && alpha.y() >= 1.0 && alpha.z() >= 1.0)
            {
                break;
            }

            t += dt;
        }
    }
    pixel_array[dy * max_x + dx] = color;
}


/**
 * VolumeRenderer class
 */

VolumeRenderer::VolumeRenderer()
    : main_camera(nullptr)
    , classifier(nullptr)
    , geometry(nullptr)
    , lights(nullptr)
{
}

VolumeRenderer::~VolumeRenderer()
{
}

void VolumeRenderer::setCamera(Camera** cam)
{
    main_camera = cam;
}

void VolumeRenderer::setLights(Light** lis, int lis_num)
{
    lights = lis;
    count_lights = lis_num;
}

void VolumeRenderer::setVolume(ImplicitGeometry** geom)
{
    geometry = geom;
}

void VolumeRenderer::setClassifier(Classifier** cls)
{
    classifier = cls;

}

void VolumeRenderer::renderFrontToBack(Eigen::Vector3f* pixel_array, int res_x, int res_y, float ambient, float shininess, float dt)
{
    int tx = CUDA_BLOCK_THREADS_X;
    int ty = CUDA_BLOCK_THREADS_Y;
    dim3 blocks(res_x / tx + 1, res_y / ty + 1);
    dim3 threads(tx, ty);
    rayIntegral<<<blocks, threads>>>(pixel_array, geometry, main_camera, lights, count_lights, classifier, ambient, shininess, dt);
    checkCudaErrors(hipDeviceSynchronize());
}
