#include "hip/hip_runtime.h"
#pragma once
#include "bbox.hpp"


/**
 * AABB class
 */

__host__ __device__ AABB::AABB()
    : lb(0, 0, 0)
    , ub(0, 0, 0)
{
}
    
__host__ __device__ AABB::AABB(float lb_x, float lb_y, float lb_z, float ub_x, float ub_y, float ub_z)
{
    lb = Eigen::Vector3f(lb_x, lb_y, lb_z);
    ub = Eigen::Vector3f(ub_x, ub_y, ub_z);
}

__host__ __device__ AABB::AABB(Eigen::Vector3f lb, Eigen::Vector3f ub)
    : lb(lb)
    , ub(ub)
{
}

__host__ __device__ AABB::AABB(const Eigen::Vector3f& pos, float radius)
{
    Eigen::Vector3f r(radius, radius, radius);
    lb = pos - r;
    ub = pos + r;
}

__host__ __device__ AABB::AABB(const Eigen::Vector3f& v1, const Eigen::Vector3f& v2, const Eigen::Vector3f& v3)
{
    lb = v1.cwiseMin(v2).cwiseMin(v3);
    ub = v1.cwiseMax(v2).cwiseMax(v3);
}

__host__ __device__ AABB::AABB(const AABB& a, const AABB& b)
{
    lb = Eigen::Vector3f(a.lb.cwiseMin(b.lb));
    ub = Eigen::Vector3f(a.ub.cwiseMax(b.ub));
}

__host__ __device__ Eigen::Vector3f AABB::getCenter() const
{
    return (lb + ub) / 2;
}

__host__ __device__ Eigen::Vector3f AABB::getSize() const
{
    return ub - lb;
}

__host__ __device__ float AABB::getDist(int c) const
{
    return ub[c] - lb[c];
}

__host__ __device__ float AABB::getVolume() const
{
    return getDist(2) * getDist(1) * getDist(0);
}

__host__ __device__ bool AABB::isOverlap(const AABB& a) const
{
    return ((a.lb[0] >= this->lb[0] && a.lb[0] <= this->ub[0]) || (this->lb[0] >= a.lb[0] && this->lb[0] <= a.ub[0])) &&
        ((a.lb[1] >= this->lb[1] && a.lb[1] <= this->ub[1]) || (this->lb[1] >= a.lb[1] && this->lb[1] <= a.ub[1])) &&
        ((a.lb[2] >= this->lb[2] && a.lb[2] <= this->ub[2]) || (this->lb[2] >= a.lb[2] && this->lb[2] <= a.ub[2]));

}

__host__ __device__ float AABB::diagonalLength() const
{
    return (ub - lb).norm();
}

__host__ __device__ bool AABB::rayIntersection(const Ray& ray, float& t_in, float& t_out) const
{
    float dir_frac_x = (ray.direction[0] == 0.0) ? 1.0e32 : 1.0f / ray.direction[0];
    float dir_frac_y = (ray.direction[1] == 0.0) ? 1.0e32 : 1.0f / ray.direction[1];
    float dir_frac_z = (ray.direction[2] == 0.0) ? 1.0e32 : 1.0f / ray.direction[2];

    float tx1 = (lb[0] - ray.origin[0]) * dir_frac_x;
    float tx2 = (ub[0] - ray.origin[0]) * dir_frac_x;
    float ty1 = (lb[1] - ray.origin[1]) * dir_frac_y;
    float ty2 = (ub[1] - ray.origin[1]) * dir_frac_y;
    float tz1 = (lb[2] - ray.origin[2]) * dir_frac_z;
    float tz2 = (ub[2] - ray.origin[2]) * dir_frac_z;

    t_in = max(max(min(tx1, tx2), min(ty1, ty2)), min(tz1, tz2));
    t_out = min(min(max(tx1, tx2), max(ty1, ty2)), max(tz1, tz2));

    /* When t_out < 0 and the ray is intersecting with AABB, the whole AABB is behind us */
    if (t_out < 0)
    {
        return false;
    }

    return t_out >= t_in;
}
